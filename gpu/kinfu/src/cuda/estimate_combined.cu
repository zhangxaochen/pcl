#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 * 
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

//#include <pcl/gpu/utils/device/block.hpp>
//#include <pcl/gpu/utils/device/funcattrib.hpp>
#include "device.hpp"

//zhangxaochen:
#include <assert.h>

namespace pcl
{
  namespace device
  {
    typedef double float_type;

    template<int CTA_SIZE_, typename T>
    static __device__ __forceinline__ void reduce(volatile T* buffer)
    {
      int tid = Block::flattenedThreadId();
      T val =  buffer[tid];

      if (CTA_SIZE_ >= 1024) { if (tid < 512) buffer[tid] = val = val + buffer[tid + 512]; __syncthreads(); }
      if (CTA_SIZE_ >=  512) { if (tid < 256) buffer[tid] = val = val + buffer[tid + 256]; __syncthreads(); }
      if (CTA_SIZE_ >=  256) { if (tid < 128) buffer[tid] = val = val + buffer[tid + 128]; __syncthreads(); }
      if (CTA_SIZE_ >=  128) { if (tid <  64) buffer[tid] = val = val + buffer[tid +  64]; __syncthreads(); }

      if (tid < 32)
      {
        if (CTA_SIZE_ >=   64) { buffer[tid] = val = val + buffer[tid +  32]; }
        if (CTA_SIZE_ >=   32) { buffer[tid] = val = val + buffer[tid +  16]; }
        if (CTA_SIZE_ >=   16) { buffer[tid] = val = val + buffer[tid +   8]; }
        if (CTA_SIZE_ >=    8) { buffer[tid] = val = val + buffer[tid +   4]; }
        if (CTA_SIZE_ >=    4) { buffer[tid] = val = val + buffer[tid +   2]; }
        if (CTA_SIZE_ >=    2) { buffer[tid] = val = val + buffer[tid +   1]; }
      }
    }

    struct Combined
    {
      enum
      {
        CTA_SIZE_X = 32,
        CTA_SIZE_Y = 8,
        CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y
      };


      Mat33 Rcurr;
      float3 tcurr;

      PtrStep<float> vmap_curr;
      PtrStep<float> nmap_curr;

      //sunguofei
      PtrStep<float> vmap_contour;
      PtrStep<float> vmap_candidate;
      PtrStep<float> nmap_candidate;

      Mat33 Rprev_inv;
      float3 tprev;

      Intr intr;

      PtrStep<float> vmap_g_prev;
      PtrStep<float> nmap_g_prev;

      float distThres;
      float angleThres;

      int cols;
      int rows;

      //sunguofei
      int rows_contour;
      double weight;

      mutable PtrStep<float_type> gbuf;

      __device__ __forceinline__ bool
      search (int x, int y, float3& n, float3& d, float3& s) const
      {
//#ifdef GCOO_2_CAMCOO //g2c
#if 0
          //�� GCOO_2_CAMCOO, ���� v/nmap_g_prev ���ڵ�ǰ�ӵ��»�õ�, ���Դ˴������� kinfu.orig ������ cam_curr->world->cam_prev ����ת��
          float3 ncurr;
          ncurr.x = nmap_curr.ptr(y)[x];
          if(isnan(ncurr.x))
              return false;

          ncurr.y = nmap_curr.ptr(y+rows)[x];
          ncurr.z = nmap_curr.ptr(y+2*rows)[x];

          float3 vcurr;
          vcurr.x = vmap_curr.ptr(y)[x];
          vcurr.y = vmap_curr.ptr(y+rows)[x];
          vcurr.z = vmap_curr.ptr(y+2*rows)[x];

          //���ﲻ��Ҫ�� kinfu.orig ������� ukr ��������, ֱ���� [x,y]
          float3 nprev_g;
          nprev_g.x = nmap_g_prev.ptr(y)[x];
          if(isnan(nprev_g.x))
              return false;

          nprev_g.y = nmap_g_prev.ptr(y+rows)[x];
          nprev_g.z = nmap_g_prev.ptr(y+2*rows)[x];

          float3 vprev_g;
          vprev_g.x = vmap_g_prev.ptr(y)[x];
          vprev_g.y = vmap_g_prev.ptr(y+rows)[x];
          vprev_g.z = vmap_g_prev.ptr(y+2*rows)[x];

          //������ Rcurr.inv, ���� kernel ����Ч, ������ʱ����
          Mat33 Rcurr_inv = Rcurr; //������ת��, �������
          Rcurr_inv.data[0].y = Rcurr.data[1].x; //21->12
          Rcurr_inv.data[0].z = Rcurr.data[2].x; //31->13
          Rcurr_inv.data[1].z = Rcurr.data[2].y; //32->23

          Rcurr_inv.data[1].x = Rcurr.data[0].y; //12->21
          Rcurr_inv.data[2].x = Rcurr.data[0].z; //13->31
          Rcurr_inv.data[2].y = Rcurr.data[1].z; //23->32

          float3 vprev_cc = Rcurr_inv * (vprev_g - tcurr); //cc: curr-cam_coo
          float3 nprev_cc = Rcurr_inv * nprev_g;

          float dist = norm (vcurr - vprev_cc);
          if (dist > distThres)
              return false;

          float sine = norm (cross (ncurr, nprev_cc));
          if (sine >= angleThres)
              return false;

          n = nprev_cc;
          d = vprev_cc;
          s = vcurr; //��������ֵ�᲻�ᵼ�´���? ��ʱ��Ϊ���� ��δ���ԡ�
          return true;

#else //kinfu.orig
        float3 ncurr;
        ncurr.x = nmap_curr.ptr (y)[x];

        if (isnan (ncurr.x))
          return (false);

        float3 vcurr;
        vcurr.x = vmap_curr.ptr (y       )[x];
        vcurr.y = vmap_curr.ptr (y + rows)[x];
        vcurr.z = vmap_curr.ptr (y + 2 * rows)[x];

        float3 vcurr_g = Rcurr * vcurr + tcurr;

        float3 vcurr_cp = Rprev_inv * (vcurr_g - tprev);         // prev camera coo space

        int2 ukr;         //projection
        ukr.x = __float2int_rn (vcurr_cp.x * intr.fx / vcurr_cp.z + intr.cx);      //4
        ukr.y = __float2int_rn (vcurr_cp.y * intr.fy / vcurr_cp.z + intr.cy);                      //4

        //����: �ⲿ���� Rprev_inv == Rcurr_inv ֮��, �Ƿ����ﲻ�����¼���?   //2016-5-29 15:19:37
        //��: ��ȷ��
        //ukr.x = x;
        //ukr.y = y;

        if (ukr.x < 0 || ukr.y < 0 || ukr.x >= cols || ukr.y >= rows || vcurr_cp.z < 0)
          return (false);

        float3 nprev_g;
        nprev_g.x = nmap_g_prev.ptr (ukr.y)[ukr.x];

        if (isnan (nprev_g.x))
          return (false);

        float3 vprev_g;
        vprev_g.x = vmap_g_prev.ptr (ukr.y       )[ukr.x];
        vprev_g.y = vmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        vprev_g.z = vmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float dist = norm (vprev_g - vcurr_g);
        if (dist > distThres)
          return (false);

        ncurr.y = nmap_curr.ptr (y + rows)[x];
        ncurr.z = nmap_curr.ptr (y + 2 * rows)[x];

        float3 ncurr_g = Rcurr * ncurr;

        nprev_g.y = nmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        nprev_g.z = nmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float sine = norm (cross (ncurr_g, nprev_g));

        if (sine >= angleThres)
          return (false);
        n = nprev_g;
        d = vprev_g;
        s = vcurr_g;
        return (true);
#endif //GCOO_2_CAMCOO
      }

      //sunguofei---contour cue //zhangxaochen: 1. ԭ��ֻ�����, ƥ���Ӧ������ cu ֮������; 2. �Ը�Ϊ: ���� !qnan �ж� //2016-3-27 17:05:43
      __device__ __forceinline__ bool
      search_contourCue (int x, int y, float3& n, float3& d, float3& s) const
      {
        float3 vcurr;
        vcurr.x = vmap_contour.ptr (y       )[x];
        if(isnan(vcurr.x)) //cores_v_curr_new @kinfu.cpp ����qnan���, ��˴˴�Ҫ����
            return (false);
            ;
        vcurr.y = vmap_contour.ptr (y + rows_contour)[x];
        vcurr.z = vmap_contour.ptr (y + 2 * rows_contour)[x];

        float3 vcurr_g = Rcurr * vcurr + tcurr;

        float3 nprev_g;
        nprev_g.x = nmap_candidate.ptr (y)[x];

        if (isnan (nprev_g.x))
          return (false);

        float3 vprev_g;
        vprev_g.x = vmap_candidate.ptr (y       )[x];
        vprev_g.y = vmap_candidate.ptr (y + rows_contour)[x];
        vprev_g.z = vmap_candidate.ptr (y + 2 * rows_contour)[x];

        nprev_g.y = nmap_candidate.ptr (y + rows_contour)[x];
        nprev_g.z = nmap_candidate.ptr (y + 2 * rows_contour)[x];

        n = nprev_g;
        d = vprev_g;
        s = vcurr_g;
        return (true);
      }

      __device__ __forceinline__ void
      operator () () const
      {
        int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        float3 n, d, s;
        bool found_coresp = false;

        if (x < cols && y < rows+rows_contour)
        {
            if (y<rows)
                found_coresp = search (x, y, n, d, s);
            else
            {
                found_coresp = search_contourCue (x, y-rows, n, d, s);
                n=n*weight;
            }
        }

        float row[7];

        if (found_coresp)
        {
          *(float3*)&row[0] = cross (s, n);
          *(float3*)&row[3] = n;
          row[6] = dot (n, d - s);
        }
        else
          row[0] = row[1] = row[2] = row[3] = row[4] = row[5] = row[6] = 0.f;

        __shared__ float_type smem[CTA_SIZE];
        int tid = Block::flattenedThreadId ();

        int shift = 0;
        for (int i = 0; i < 6; ++i)        //rows
        {
          #pragma unroll
          for (int j = i; j < 7; ++j)          // cols + b
          {
            __syncthreads ();
            smem[tid] = row[i] * row[j];
            __syncthreads ();

            reduce<CTA_SIZE>(smem);

            if (tid == 0)
              gbuf.ptr (shift++)[blockIdx.x + gridDim.x * blockIdx.y] = smem[0];
          }
        }
      }
    };

    __global__ void
    combinedKernel (const Combined cs) 
    {
      cs ();
    }

    //@brief Ϊ����֤ combinedKernel �����Ƿ���ÿ�� kernel thread ����һ�� CombinedCCW ʵ�����ֲ�����д CombinedCCW-copy-ctor
    struct CombinedCCWLog{
        CombinedCCWLog(){
            printf("CombinedCCWLog-ctor\n");
        }
    };

    __device__ int g_contCorrespCounter = 0;

    //@author zhangxaochen
    //@brief CCW, i.e., Contour-Cue increased Weight
    struct CombinedCCW : public Combined{
        PtrStepSz<unsigned char> contourMsk;
        float contWeight;
        int contCorrespCounter;
        //CombinedCCWLog tmpLog; //����� default-ctor ǰ����� log�� copy-ctor û��

        CombinedCCW() : contWeight(-1), contCorrespCounter(0)
        {
            //printf("CombinedCCW-default-ctor~\n"); //Ĭ�Ϲ���, estimateCombined(host)�е��ã���ÿ���̹߳���һ��
                                                   //def-ctor, copy-ctor �ж����ⲻ��ʼ�� CombinedCCWLog�� �����乹�캯����Ȼ�������ˣ�
                                                   //copy-ctor ���ÿ���̹߳���һ�Ρ�����http://www.evernote.com/l/AY_3ITYZDqhPuZ4RuNiQzt6SjXKq9pcdvqw/
        }
        //CombinedCCW(const CombinedCCW &other)
        //{
        //    Rcurr = other.Rcurr;
        //    tcurr = other.tcurr;
        //    vmap_curr = other.vmap_curr;
        //    nmap_curr = other.nmap_curr;
        //    vmap_contour = other.vmap_contour;
        //    vmap_candidate = other.vmap_candidate;
        //    nmap_candidate = other.nmap_candidate;
        //    Rprev_inv = other.Rprev_inv;
        //    tprev = other.tprev;
        //    intr = other.intr;
        //    vmap_g_prev = other.vmap_g_prev;
        //    nmap_g_prev = other.nmap_g_prev;
        //    distThres = other.distThres;
        //    angleThres = other.angleThres;
        //    cols = other.cols;
        //    rows = other.rows;
        //    rows_contour = other.rows_contour;
        //    weight = other.weight;
        //    gbuf = other.gbuf;

        //    printf("CombinedCCW-copy-ctor~\n"); //
        //}

        __device__ __forceinline__ void
        operator () () const
        {
            int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
            int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

            float3 n, d, s;
            bool found_coresp = false;

            if (x < cols && y < rows)
                found_coresp = search (x, y, n, d, s);

            float row[7];

            if (found_coresp)
            {
                if(contWeight > 0){
                    unsigned char mskVal = contourMsk.ptr(y)[x];
                    if(mskVal != 0){
                        //atomicAdd(&contCorrespCounter, 1); //Error: unsupported operation, ���� __device__ ԭ�� ������δ�����
                        atomicAdd(&g_contCorrespCounter, 1);
                        n = n * contWeight;
                        //printf("mskVal: %d, [%f, %f, %f]\n", mskVal, n.x, n.y, n.z);
                    }
                }

                *(float3*)&row[0] = cross (s, n);
                *(float3*)&row[3] = n;
                row[6] = dot (n, d - s);
            }
            else
                row[0] = row[1] = row[2] = row[3] = row[4] = row[5] = row[6] = 0.f;

            __shared__ float_type smem[CTA_SIZE];
            int tid = Block::flattenedThreadId ();

            int shift = 0;
            for (int i = 0; i < 6; ++i)        //rows
            {
#pragma unroll
                for (int j = i; j < 7; ++j)          // cols + b
                {
                    __syncthreads ();
                    smem[tid] = row[i] * row[j];
                    __syncthreads ();

                    reduce<CTA_SIZE>(smem);

                    if (tid == 0)
                        gbuf.ptr (shift++)[blockIdx.x + gridDim.x * blockIdx.y] = smem[0];
                }
            }
        }//operator () () const
    };//struct CombinedCCW

    //@author zhangxaochen
    __global__ void
    combinedKernelCCW (const CombinedCCW cs) 
    {
        //g_contCorrespCounter = 0; //���С���Զ��0
        cs ();
    }

    struct TranformReduction
    {
      enum
      {
        CTA_SIZE = 512,
        STRIDE = CTA_SIZE,

        B = 6, COLS = 6, ROWS = 6, DIAG = 6,
        UPPER_DIAG_MAT = (COLS * ROWS - DIAG) / 2 + DIAG,
        TOTAL = UPPER_DIAG_MAT + B,

        GRID_X = TOTAL
      };

      PtrStep<float_type> gbuf;
      int length;
      mutable float_type* output;

      __device__ __forceinline__ void
      operator () () const
      {
        const float_type *beg = gbuf.ptr (blockIdx.x);
        const float_type *end = beg + length;

        int tid = threadIdx.x;

        float_type sum = 0.f;
        for (const float_type *t = beg + tid; t < end; t += STRIDE)
          sum += *t;

        __shared__ float_type smem[CTA_SIZE];

        smem[tid] = sum;
        __syncthreads ();

		reduce<CTA_SIZE>(smem);

        if (tid == 0)
          output[blockIdx.x] = smem[0];
      }
    };

    __global__ void
    TransformEstimatorKernel2 (const TranformReduction tr) 
    {
      tr ();
    }
  }//namespace device
}//namespace pcl


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::estimateCombined (const Mat33& Rcurr, const float3& tcurr, 
                               const MapArr& vmap_curr, const MapArr& nmap_curr, 
                               const Mat33& Rprev_inv, const float3& tprev, const Intr& intr,
                               const MapArr& vmap_g_prev, const MapArr& nmap_g_prev, 
                               float distThres, float angleThres,
                               DeviceArray2D<float_type>& gbuf, DeviceArray<float_type>& mbuf,
                               float_type* matrixA_host, float_type* vectorB_host)
{
  int cols = vmap_curr.cols ();
  int rows = vmap_curr.rows () / 3;

  Combined cs;

  cs.Rcurr = Rcurr;
  cs.tcurr = tcurr;

  cs.vmap_curr = vmap_curr;
  cs.nmap_curr = nmap_curr;

  cs.Rprev_inv = Rprev_inv;
  cs.tprev = tprev;

  cs.intr = intr;

  cs.vmap_g_prev = vmap_g_prev;
  cs.nmap_g_prev = nmap_g_prev;

  cs.distThres = distThres;
  cs.angleThres = angleThres;

  cs.cols = cols;
  cs.rows = rows;
  //sunguofei
  cs.rows_contour = 0;

//////////////////////////////

  dim3 block (Combined::CTA_SIZE_X, Combined::CTA_SIZE_Y);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  mbuf.create (TranformReduction::TOTAL);
  if (gbuf.rows () != TranformReduction::TOTAL || gbuf.cols () < (int)(grid.x * grid.y))
    gbuf.create (TranformReduction::TOTAL, grid.x * grid.y);

  cs.gbuf = gbuf;

  combinedKernel<<<grid, block>>>(cs);
  cudaSafeCall ( hipGetLastError () );
  //cudaSafeCall(hipDeviceSynchronize());

  //printFuncAttrib(combinedKernel);

  TranformReduction tr;
  tr.gbuf = gbuf;
  tr.length = grid.x * grid.y;
  tr.output = mbuf;

  TransformEstimatorKernel2<<<TranformReduction::TOTAL, TranformReduction::CTA_SIZE>>>(tr);
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());

  float_type host_data[TranformReduction::TOTAL];
  mbuf.download (host_data);

  int shift = 0;
  for (int i = 0; i < 6; ++i)  //rows
    for (int j = i; j < 7; ++j)    // cols + b
    {
      float_type value = host_data[shift++];
      if (j == 6)       // vector b
        vectorB_host[i] = value;
      else
        matrixA_host[j * 6 + i] = matrixA_host[i * 6 + j] = value;
    }
}



//@author zhangxaochen
//@brief ��ԭ estimateCombined �������� �ٵ�ǰ֡contourMsk����contourȨ�ز���contWeight(>1)
void
pcl::device::estimateCombined (const Mat33 &Rcurr, const float3 &tcurr, 
                               const MapArr &vmap_curr, const MapArr &nmap_curr, 
                               const Mat33 &Rprev_inv, const float3 &tprev, const Intr &intr,
                               const MapArr &vmap_g_prev, const MapArr &nmap_g_prev, 
                               float distThres, float angleThres,
                               DeviceArray2D<float_type> &gbuf, DeviceArray<float_type> &mbuf,
                               float_type *matrixA_host, float_type *vectorB_host,
                               DeviceArray2D<unsigned char> &contourMsk, float contWeight)
{
  int cols = vmap_curr.cols ();
  int rows = vmap_curr.rows () / 3;

  //zhangxaochen:
  assert(contWeight > 0);
  pcl::device::CombinedCCW cs; 
  cs.contWeight = contWeight;
  cs.contourMsk = contourMsk;

  cs.Rcurr = Rcurr;
  cs.tcurr = tcurr;

  cs.vmap_curr = vmap_curr;
  cs.nmap_curr = nmap_curr;

  cs.Rprev_inv = Rprev_inv;
  cs.tprev = tprev;

  cs.intr = intr;

  cs.vmap_g_prev = vmap_g_prev;
  cs.nmap_g_prev = nmap_g_prev;

  cs.distThres = distThres;
  cs.angleThres = angleThres;

  cs.cols = cols;
  cs.rows = rows;
  //sunguofei
  cs.rows_contour = 0;

//////////////////////////////

  dim3 block (Combined::CTA_SIZE_X, Combined::CTA_SIZE_Y);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  mbuf.create (TranformReduction::TOTAL);
  if (gbuf.rows () != TranformReduction::TOTAL || gbuf.cols () < (int)(grid.x * grid.y))
    gbuf.create (TranformReduction::TOTAL, grid.x * grid.y);

  cs.gbuf = gbuf;

  //g_contCorrespCounter = 0; //���, �޷����³�ʼ�����㣬��������
                              //��Ӧ�� host code ���������ҵ��ʴ�http://stackoverflow.com/questions/19185484/passing-value-from-device-memory-as-kernel-parameter-in-cuda
                            //�״ε���ֵ 299, ������������
  int contCorrespCounter = 0;

  hipMemcpyToSymbol(HIP_SYMBOL(g_contCorrespCounter), &contCorrespCounter, sizeof(int)); //����, invalid device symbol
  //hipMemset(&g_contCorrespCounter, 0, sizeof(int)); //����, invalid argument

  hipMemcpyFromSymbol(&contCorrespCounter, HIP_SYMBOL(g_contCorrespCounter), sizeof(int));
  printf("g_contCorrespCounter~: %d\n", contCorrespCounter);

  combinedKernelCCW<<<grid, block>>>(cs);

  //sync(); //����
  //hipMemcpy(&contCorrespCounter, &g_contCorrespCounter, sizeof(int), hipMemcpyDeviceToHost); //�������� hipMemcpyFromSymbol
  hipMemcpyFromSymbol(&contCorrespCounter, HIP_SYMBOL(g_contCorrespCounter), sizeof(int));
  printf("g_contCorrespCounter: %d\n", contCorrespCounter);
  cudaSafeCall ( hipGetLastError () );
  //cudaSafeCall(hipDeviceSynchronize());

  //printFuncAttrib(combinedKernel);

  TranformReduction tr;
  tr.gbuf = gbuf;
  tr.length = grid.x * grid.y;
  tr.output = mbuf;

  TransformEstimatorKernel2<<<TranformReduction::TOTAL, TranformReduction::CTA_SIZE>>>(tr);
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());

  float_type host_data[TranformReduction::TOTAL];
  mbuf.download (host_data);

  int shift = 0;
  for (int i = 0; i < 6; ++i)  //rows
    for (int j = i; j < 7; ++j)    // cols + b
    {
      float_type value = host_data[shift++];
      if (j == 6)       // vector b
        vectorB_host[i] = value;
      else
        matrixA_host[j * 6 + i] = matrixA_host[i * 6 + j] = value;
    }
}//estimateCombin (CCW)


//sunguofei
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::estimateCombined (const Mat33& Rcurr, const float3& tcurr, 
                               const MapArr& vmap_curr, const MapArr& nmap_curr, 
                               const MapArr& vmap_contour, const MapArr& vmap_candidate, const MapArr& nmap_candidate,
                               const Mat33& Rprev_inv, const float3& tprev, const Intr& intr,
                               const MapArr& vmap_g_prev, const MapArr& nmap_g_prev, 
                               float distThres, float angleThres,
                               DeviceArray2D<float_type>& gbuf, DeviceArray<float_type>& mbuf,
                               float_type* matrixA_host, float_type* vectorB_host,double weight)
{
  int cols = vmap_curr.cols ();
  int rows = vmap_curr.rows () / 3;
  int rows_contour = vmap_contour.rows () / 3;

  Combined cs;

  cs.Rcurr = Rcurr;
  cs.tcurr = tcurr;

  cs.vmap_curr = vmap_curr;
  cs.nmap_curr = nmap_curr;

  cs.vmap_contour = vmap_contour;
  cs.vmap_candidate = vmap_candidate;
  cs.nmap_candidate = nmap_candidate;

  cs.Rprev_inv = Rprev_inv;
  cs.tprev = tprev;

  cs.intr = intr;

  cs.vmap_g_prev = vmap_g_prev;
  cs.nmap_g_prev = nmap_g_prev;

  cs.distThres = distThres;
  cs.angleThres = angleThres;

  cs.cols = cols;
  cs.rows = rows;
  cs.rows_contour = rows_contour;
  cs.weight=weight;

//////////////////////////////

  dim3 block (Combined::CTA_SIZE_X, Combined::CTA_SIZE_Y);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows+rows_contour, block.y);

  mbuf.create (TranformReduction::TOTAL);
  if (gbuf.rows () != TranformReduction::TOTAL || gbuf.cols () < (int)(grid.x * grid.y))
    gbuf.create (TranformReduction::TOTAL, grid.x * grid.y);

  cs.gbuf = gbuf;

  combinedKernel<<<grid, block>>>(cs);
  cudaSafeCall ( hipGetLastError () );
  //cudaSafeCall(hipDeviceSynchronize());

  //printFuncAttrib(combinedKernel);

  TranformReduction tr;
  tr.gbuf = gbuf;
  tr.length = grid.x * grid.y;
  tr.output = mbuf;

  TransformEstimatorKernel2<<<TranformReduction::TOTAL, TranformReduction::CTA_SIZE>>>(tr);
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());

  float_type host_data[TranformReduction::TOTAL];
  mbuf.download (host_data);

  int shift = 0;
  for (int i = 0; i < 6; ++i)  //rows
    for (int j = i; j < 7; ++j)    // cols + b
    {
      float_type value = host_data[shift++];
      if (j == 6)       // vector b
        vectorB_host[i] = value;
      else
        matrixA_host[j * 6 + i] = matrixA_host[i * 6 + j] = value;
    }
}
